#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i ++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out, *a2;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    a2  = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    gpuErrchk( hipMalloc((void**)&d_a, sizeof(float) * N) );
    gpuErrchk( hipMalloc((void**)&d_b, sizeof(float) * N) );
    gpuErrchk( hipMalloc((void**)&d_out, sizeof(float) * N) );

    // Transfer data from host to device memory
    gpuErrchk( hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice ));
    gpuErrchk( hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice) );

    // Executing kernel 
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    gpuErrchk( hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(a2, d_a, sizeof(float) * N, hipMemcpyDeviceToHost) );

    printf("out[0] = %f\n", out[0]);
    printf("a from device[0] = %f\n", a2[0]);
    printf("Expected answer is 1+2=3...\n");
    // Verification
    for(int i = 0; i < N; i++){
        if ( fabs(out[i] - a[i] - b[i]) > MAX_ERR ) {
            printf(" %f %f %f\n",out[i],a[i],b[i]);
        }
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("SUCCESS!\n");
    
    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
